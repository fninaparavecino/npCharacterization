
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <time.h>
#include <sys/time.h>
using namespace std;
//#define ROWS 1024
//#define COLS 1024
__device__ int parentIdx[1024];

__global__ void childKernel(int* A, int *B, int *C, int parentIdxVar)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	C[parentIdxVar+idx] = A[parentIdxVar+idx] + B[parentIdxVar+idx];
}
__global__ void parentKernel(int* A, int *B, int *C, int rows, int cols)
{
	int idx = blockIdx.x *blockDim.x + threadIdx.x;
	if(A[idx*cols] == 1)
	{	
		parentIdx[idx] = idx*cols;		
		childKernel<<<1, cols>>>(A, B, C, parentIdx[idx]);
	}
}
__global__ void singleKernel(int* A, int *B, int *C, int rows, int cols)
{
	int idx = blockIdx.x *blockDim.x + threadIdx.x;
	if(A[idx*cols] == 1)
	{	
		for(int i=0; i < cols; i++)
			C[idx*cols+i] = A[idx*cols+i]+B[idx*cols+i];
	}
}
void printOutput(int *A, int rows, int cols)
{
	for(int i=0; i < rows; i++)
	{
		for(int j=0; j < cols; j++){
			printf("%d ", A[i*cols+j]);
		}
		printf("\n");
	}
}
bool check(int *c1, int *c2, int rows, int cols){
	
	bool same = true;
	for(int i=0; i < rows; i++)
	{
		for(int j=0; j < cols; j++){
			if(c1[i*cols+j] != c2[i*cols+j]){
				same = false;
				break;
			}				
		}
		if (!same)
			break;
	}
	return same;
}
double getWallTime(){
        struct timeval time;
        if(gettimeofday(&time,NULL)){
                printf("Error getting time\n");
                return 0;
        }
        return (double)time.tv_sec + (double)time.tv_usec * .000001;
}
int main(int argC, char** argV)
{
	printf("NP - Characterization\n");
	int ROWS = 1024, COLS = 1024;
	for(int i=1; i<argC; i++)
	{
		if(strcmp(argV[i], "-size") == 0)
		{
			if(i+1 < argC)
			{
				ROWS = atoi(argV[i+1]);
				COLS = ROWS;
				printf("%d %d\n", ROWS, COLS);
				if(ROWS < 1)
				{
					cerr << "Size must be greater than 0." << endl;
					exit(1);
				}
			}
			else
			{
				printf("Error...\n");
				exit(1);
			}
		}
		else if(strcmp(argV[i], "-h") == 0 || strcmp(argV[i], "--help") == 0)
		{
			cout << "Usage: " << argV[0] << " [OPTIONS] -size <number>" << endl;
			cout << "  -h, --help            Display this information and exit." << endl;

			exit(0);
		}
		else
		{
			cerr << "Did not recognize '" << argV[i] << "'. Try '" << argV[0]
				<< " --help' for additional information." << endl;
			exit(1);
		}
	}
	
	printf("NP Case2: [%d x %d]\n", ROWS, COLS);
	int *a = (int*) malloc(ROWS*COLS*sizeof(int));
	int *b = (int*) malloc(ROWS*COLS*sizeof(int));
	int *c = (int*) malloc(ROWS*COLS*sizeof(int));
	for (int i=0; i<ROWS; i++){
		for(int j=0; j<COLS; j++){
			if(i%8 == 0){
				a[i*COLS+j] = 1;
				b[i*COLS+j] = 2;
			}
		}
	}
	// Sequential
	double wallS0, wallS1;
	wallS0 = getWallTime();
	int *cHost = (int*)malloc(ROWS*COLS*sizeof(int));
	for(int i=0; i<ROWS; i++){
		if(a[i*COLS] == 1)
			for(int j=0; j<COLS; j++){
				cHost[i*COLS+j] = a[i*COLS+j] + b[i*COLS+j];
			}
	}
	wallS1 = getWallTime();
	printf("\tSequential Job Time: %f ms\n", (wallS1-wallS0)*1000);
	// Time variables
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
		
	int *devA;
	int *devB;
	hipMalloc((void**)&devA, ROWS*COLS*sizeof(int));
	hipMalloc((void**)&devB, ROWS*COLS*sizeof(int));
		
	//Copying [A] and [B] from host memory to device memory.
	hipMemcpy(devA, a, ROWS*COLS*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(devB, b, ROWS*COLS*sizeof(int), hipMemcpyHostToDevice);
		
	// Single Kernel **********************************************************
	int *devC2;
	hipMalloc((void**)&devC2, ROWS*COLS*sizeof(int));	
	hipMemcpy(devC2, c, ROWS*COLS*sizeof(int), hipMemcpyHostToDevice);
	hipEventRecord(start, 0);
	singleKernel<<<1,ROWS>>>(devA, devB, devC2, ROWS, COLS);
	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	
	//Display time
	hipEventElapsedTime(&time, start, stop);
	printf("\tParallel Job time single kernel: %.2f ms\n", time);
	
	//Retrieve results from device
	hipMemcpy(c, devC2, ROWS*COLS*sizeof(int), hipMemcpyDeviceToHost);
	//Verify correctness	
	check(c, cHost, ROWS, COLS) ? printf("Results are correct.\n") : printf("ERROR! Results are not the same");

	// NP Case ****************************************************************
	int *devC;
	hipMalloc((void**)&devC, ROWS*COLS*sizeof(int));
	hipMemcpy(devC, c, ROWS*COLS*sizeof(int), hipMemcpyHostToDevice);
	hipEventRecord(start, 0);
	parentKernel<<<1,ROWS>>>(devA, devB, devC, ROWS, COLS);
	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	
	//Display time
	hipEventElapsedTime(&time, start, stop);
	printf("\tParallel Job time: %.2f ms\n", time);
	
	//Retrieve results from device
	hipMemcpy(c, devC, ROWS*COLS*sizeof(int), hipMemcpyDeviceToHost);
	//Verify correctness	
	check(c, cHost, ROWS, COLS) ? printf("Results are correct.\n") : printf("ERROR! Results are not the same");
}
