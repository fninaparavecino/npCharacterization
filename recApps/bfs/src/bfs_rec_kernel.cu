#include "hip/hip_runtime.h"
#ifndef __BFS_REC_KERNEL__
#define __BFS_REC_KERNEL__

#define MAX_LEVEL 9999
#define MAXDIMGRID 65535
#define MAX_THREAD_PER_BLOCK 1024

#define WARP_SIZE 32
#define SHM_BUFF_SIZE 256

#define GPU_PROFILE

#ifdef GPU_PROFILE
// records the number of kerbel calls performed
__device__ unsigned nested_calls = 0;

__global__ void gpu_statistics(unsigned solution){
	printf("====> GPU #%u - number of kernel calls:%u\n",solution, nested_calls);
}

__global__ void reset_gpu_statistics(){
	nested_calls = 0;
}
#endif

//#if (CONSOLIDATE_LEVEL==2)

__device__ unsigned int tmp_buffer[GM_BUFF_SIZE];
__device__ unsigned int tmp_idx;

//#endif

__global__ void gpu_print(unsigned int *idx)
{
	printf("index: %d\n", *idx);

}

__device__ unsigned int gm_idx_pool[2000][1];

// iterative, flat BFS traversal (note: synchronization-free implementation)
__global__ void bfs_kernel_flat(int level, int num_nodes, int *vertexArray, int *edgeArray, int *levelArray, bool *queue_empty){
#if (PROFILE_GPU!=0)
	if (threadIdx.x+blockDim.x*blockIdx.x==0) atomicInc(&nested_calls, INF);
#endif
	unsigned tid = threadIdx.x + blockDim.x * blockIdx.x;
	for (int node = tid; node < num_nodes; node += blockDim.x * gridDim.x){
		//printf("DEBUG node in gpu flat: %d \n", node);
		if(node < num_nodes && levelArray[node] == level){
			for (int edge=vertexArray[node]; edge<vertexArray[node+1]; edge++){
				int neighbor=edgeArray[edge];
				if (levelArray[neighbor]==UNDEFINED || levelArray[neighbor]>(level+1)){
					levelArray[neighbor]=level+1;
					*queue_empty=false;
				}
			}	
		}
	}	
}

// recursive naive NFS traversal
__global__ void bfs_kernel_dp(int node, int *vertexArray, int *edgeArray, int *levelArray){
#ifdef GPU_PROFILE
	if (threadIdx.x+blockDim.x*blockIdx.x==0) atomicInc(&nested_calls, INF);
#endif

#if (STREAMS!=0)
	hipStream_t s[STREAMS];
	for (int i=0; i<STREAMS; ++i)  hipStreamCreateWithFlags(&s[i], hipStreamNonBlocking);	
#endif

	int num_children = vertexArray[node+1]-vertexArray[node];
	for (unsigned childp = threadIdx.x; childp < num_children; childp+=blockDim.x){ // may change this to use multiple blocks
		int child = edgeArray[vertexArray[node]+childp];
		int node_level = levelArray[node];
		int child_level = levelArray[child];
		if (child_level==UNDEFINED || child_level>(node_level+1)){
			unsigned old_level = atomicMin(&levelArray[child],node_level+1);
			if (old_level == child_level){
				unsigned num_grandchildren=vertexArray[child+1]-vertexArray[child];
				unsigned block_size = min(num_grandchildren, THREADS_PER_BLOCK);
#if (STREAMS!=0)
			        if (block_size!=0) bfs_kernel_dp<<<1,block_size, 0, s[threadIdx.x%STREAMS]>>>(child, vertexArray, edgeArray, levelArray);
#else
			        if (block_size!=0) bfs_kernel_dp<<<1,block_size>>>(child, vertexArray, edgeArray, levelArray);
#endif
			}
		}
	}
}

// recursive hierarchical BFS traversal
__global__ void bfs_kernel_dp_hier(int node, int *vertexArray, int *edgeArray, int *levelArray){
#if (PROFILE_GPU!=0)
	if (threadIdx.x+blockDim.x*blockIdx.x==0) atomicInc(&nested_calls, INF);
#endif

#if (STREAMS!=0)
	hipStream_t s[STREAMS];
	for (int i=0; i<STREAMS; ++i)  hipStreamCreateWithFlags(&s[i], hipStreamNonBlocking);	
#endif
	__shared__ int child;
	__shared__ int child_level;
	__shared__ unsigned num_grandchildren;
	
	int node_level = levelArray[node];
	unsigned num_children = vertexArray[node+1]-vertexArray[node];
	
	for (unsigned childp = blockIdx.x; childp < num_children; childp+=gridDim.x){
		if (threadIdx.x==0){
			child = edgeArray[vertexArray[node]+childp];
			num_grandchildren = 0; // by default, do not continue
			child_level = levelArray[child];
			if (child_level==UNDEFINED || child_level>(node_level+1)){
				unsigned old_level = atomicMin(&levelArray[child],node_level+1);
				if (old_level == child_level)
					num_grandchildren = vertexArray[child+1]-vertexArray[child];
			}
		}
		__syncthreads();
		if (num_grandchildren != 0){
			for (unsigned grandchild_p = threadIdx.x; grandchild_p < num_grandchildren; grandchild_p+=blockDim.x){
				unsigned grandchild = edgeArray[vertexArray[child]+grandchild_p];
				unsigned grandchild_level = levelArray[grandchild];
				if (grandchild_level == UNDEFINED || grandchild_level > (node_level + 2)){
					unsigned old_level = atomicMin(&levelArray[grandchild],node_level+2);
					if (old_level == grandchild_level){
						unsigned num_grandgrandchildren = vertexArray[grandchild+1]-vertexArray[grandchild];
#if (STREAMS!=0)
						if (num_grandgrandchildren!=0) 
							bfs_kernel_dp_hier<<<num_grandgrandchildren,THREADS_PER_BLOCK, 0, s[threadIdx.x%STREAMS]>>>(grandchild, vertexArray, edgeArray, levelArray);
#else 
						if (num_grandgrandchildren!=0) 
							bfs_kernel_dp_hier<<<num_grandgrandchildren,THREADS_PER_BLOCK>>>(grandchild, vertexArray, edgeArray, levelArray);
#endif
					}
				}
			}
		}
		__syncthreads();
	}
}

// prepare bfs_kernel_dp_cons
__global__ void  bfs_kernel_dp_cons_prepare(int *levelArray, unsigned int *buffer, 
													unsigned *idx, int source)
{
	levelArray[source] = 0;		// redundant
	buffer[0] = source;
	*idx = 1;
	//printf("Source : %d\n", source);
	//printf("Buffer address : %p\n", buffer);
	//printf("LevelArray address : %p\n", levelArray);
	//printf("%d\n", (buffer-0x13001b3c28));
	//printf("%d\n", sizeof(unsigned int));
}

// recursive BFS traversal with warp-level consolidation
__global__ void bfs_kernel_dp_warp_cons(int *vertexArray, int *edgeArray, int *levelArray, 
										unsigned int *queue, unsigned int queue_size,
										unsigned int *buffer, unsigned int *idx) {
	unsigned int t_idx;
	__shared__ unsigned int sh_idx[THREADS_PER_BLOCK/WARP_SIZE+1];
	__shared__ unsigned int ori_idx[THREADS_PER_BLOCK/WARP_SIZE+1];
	__shared__ unsigned int total_num_child;

	int warp_id = threadIdx.x / WARP_SIZE;
//	int warp_dim = blockDim.x / WARP_SIZE;
//	int total_warp_num = gridDim.x * warp_dim;	
	if ( threadIdx.x==0 ) {
		total_num_child = 0;
		for ( unsigned bid = blockIdx.x; bid<queue_size; bid += gridDim.x) {
			int node = queue[bid];
			total_num_child += vertexArray[node+1]-vertexArray[node];
		}
	}
	__syncthreads();

	if (threadIdx.x%WARP_SIZE==0) {
		//ori_idx[warp_id] = atomicAdd(idx, num_children);
		ori_idx[warp_id] = atomicAdd(idx, total_num_child/(WARP_SIZE/16));
		sh_idx[warp_id] = ori_idx[warp_id];
	}
	
	for (unsigned bid = blockIdx.x; bid<queue_size; bid += gridDim.x ) {
		int node = queue[bid];
		unsigned int num_children = vertexArray[node+1]-vertexArray[node];
		for (unsigned childp = threadIdx.x; childp < num_children; childp+=blockDim.x) {
			int child = edgeArray[vertexArray[node]+childp];
			unsigned node_level = levelArray[node];
			unsigned child_level = levelArray[child];
			if (child_level==UNDEFINED || child_level>(node_level+1)){
				unsigned old_level = atomicMin(&levelArray[child], node_level+1);
				t_idx = atomicInc(&sh_idx[warp_id], GM_BUFF_SIZE);
				buffer[t_idx] = child;
			}
		}
	}

	if (threadIdx.x%WARP_SIZE==0 && sh_idx[warp_id]>ori_idx[warp_id]) {
#ifdef GPU_PROFILE
		atomicInc(&nested_calls, INF);
#endif
		unsigned int size = sh_idx[warp_id]-ori_idx[warp_id];
	//	printf("Launch kernel with %d - %d = %d blocks\n", sh_idx, ori_idx, sh_idx-ori_idx);
		unsigned int block_num = 13;
		if (size<block_num) block_num = size;
		bfs_kernel_dp_warp_cons<<<block_num, THREADS_PER_BLOCK>>>(vertexArray, 
									 	edgeArray, levelArray, buffer+ori_idx[warp_id], size,
										buffer, idx);
	}

	// no post work require
}

// recursive BFS traversal with warp-level consolidation
__global__ void bfs_kernel_dp_warp_cons_unlimited(int *vertexArray, int *edgeArray, int *levelArray, 
								unsigned int *queue, unsigned int *buffer, unsigned int *idx) {
	unsigned int bid = blockIdx.x; // 1-Dimensional grid configuration
	unsigned int t_idx;
	__shared__ unsigned int sh_idx[THREADS_PER_BLOCK/WARP_SIZE+1];
	__shared__ unsigned int ori_idx[THREADS_PER_BLOCK/WARP_SIZE+1];

	int warp_id = threadIdx.x / WARP_SIZE;
//	int warp_dim = blockDim.x / WARP_SIZE;
//	int total_warp_num = gridDim.x * warp_dim;	

	int node = queue[bid];

	unsigned int num_children = vertexArray[node+1]-vertexArray[node];
	if (threadIdx.x%WARP_SIZE==0) {
		ori_idx[warp_id] = atomicAdd(idx, num_children);
		sh_idx[warp_id] = ori_idx[warp_id];
	}

	for (unsigned childp = threadIdx.x; childp < num_children; childp+=blockDim.x) {
		int child = edgeArray[vertexArray[node]+childp];
		unsigned node_level = levelArray[node];
		unsigned child_level = levelArray[child];
		if (child_level==UNDEFINED || child_level>(node_level+1)){
			unsigned old_level = atomicMin(&levelArray[child], node_level+1);
			t_idx = atomicInc(&sh_idx[warp_id], GM_BUFF_SIZE);
			buffer[t_idx] = child;
		}
	}

	if (threadIdx.x%WARP_SIZE==0 && sh_idx[warp_id]>ori_idx[warp_id]) {
#ifdef GPU_PROFILE
		atomicInc(&nested_calls, INF);
#endif
	//	printf("Launch kernel with %d - %d = %d blocks\n", sh_idx, ori_idx, sh_idx-ori_idx);
		bfs_kernel_dp_warp_cons_unlimited<<<sh_idx[warp_id]-ori_idx[warp_id], THREADS_PER_BLOCK>>>(vertexArray, 
									 	edgeArray, levelArray, buffer+ori_idx[warp_id], 
										buffer, idx);
	}

	// no post work require
}

// recursive BFS traversal with block-level consolidation
__global__ void bfs_kernel_dp_warp_malloc_cons(int *vertexArray, int *edgeArray, int *levelArray, 
								unsigned int *queue, unsigned int *buffer, unsigned int *idx) {
	unsigned int bid = blockIdx.x; // 1-Dimensional grid configuration
	unsigned int t_idx;
	__shared__ unsigned int sh_idx[THREADS_PER_BLOCK/WARP_SIZE+1];
	__shared__ unsigned int* sh_buffer[THREADS_PER_BLOCK/WARP_SIZE+1];

	int warp_id = threadIdx.x / WARP_SIZE;
//	int warp_dim = blockDim.x / WARP_SIZE;
//	int total_warp_num = gridDim.x * warp_dim;	

	int node = queue[bid];

	unsigned int num_children = vertexArray[node+1]-vertexArray[node];
	if (threadIdx.x%WARP_SIZE==0) {
		sh_buffer[warp_id] = (unsigned int*)malloc(sizeof(unsigned int)*num_children);
		sh_idx[warp_id] = 0;
	}

	for (unsigned childp = threadIdx.x; childp < num_children; childp+=blockDim.x) {
		int child = edgeArray[vertexArray[node]+childp];
		unsigned node_level = levelArray[node];
		unsigned child_level = levelArray[child];
		if (child_level==UNDEFINED || child_level>(node_level+1)){
			unsigned old_level = atomicMin(&levelArray[child], node_level+1);
			t_idx = atomicInc(&sh_idx[warp_id], GM_BUFF_SIZE);
			buffer[t_idx] = child;
		}
	}

	if (threadIdx.x%WARP_SIZE==0 && sh_idx[warp_id]>0) {
	//	printf("Launch kernel with %d - %d = %d blocks\n", sh_idx, ori_idx, sh_idx-ori_idx);
		bfs_kernel_dp_warp_malloc_cons<<<sh_idx[warp_id], THREADS_PER_BLOCK>>>(vertexArray, 
									 	edgeArray, levelArray, sh_buffer[warp_id], 
										buffer, idx);
#ifdef FORCE_SYNC
		cudaDeviceSynhronize();
		free(sh_buffer[warp_id]);
#endif	
	}

	// no post work require
}

// recursive BFS traversal with block-level consolidation
__global__ void bfs_kernel_dp_block_cons(int *vertexArray, int *edgeArray, int *levelArray, 
										unsigned int *queue, unsigned int queue_size, 
										unsigned int *buffer, unsigned int *idx) {
	unsigned int t_idx;
	__shared__ unsigned int sh_idx;
	__shared__ unsigned int ori_idx;
	__shared__ unsigned int total_num_child;
	if (threadIdx.x==0) {
		total_num_child = 0;
		for (unsigned bid = blockIdx.x; bid < queue_size; bid += gridDim.x) {
			int node = queue[bid];
			total_num_child += vertexArray[node+1]-vertexArray[node];
		}
		ori_idx = atomicAdd(idx, total_num_child);
		sh_idx = ori_idx;
	}
	__syncthreads();

	for (unsigned bid = blockIdx.x; bid < queue_size; bid += gridDim.x) {
		int node = queue[bid];
		unsigned num_children = vertexArray[node+1]-vertexArray[node];
		for (unsigned childp = threadIdx.x; childp < num_children; childp+=blockDim.x) {
			int child = edgeArray[vertexArray[node]+childp];
			unsigned child_level = levelArray[child];
			unsigned node_level = levelArray[node];
			if (child_level==UNDEFINED || child_level>(node_level+1)){
				unsigned old_level = atomicMin(&levelArray[child], node_level+1);
				t_idx = atomicInc(&sh_idx, GM_BUFF_SIZE);
				buffer[t_idx] = child;
			}
		}
	}
	__syncthreads();
	if (threadIdx.x==0 && sh_idx>ori_idx) {
#ifdef GPU_PROFILE
		atomicInc(&nested_calls, INF);
#endif
		//printf("Launch kernel with %d - %d = %d blocks\n", sh_idx, ori_idx, sh_idx-ori_idx);
		//bfs_kernel_dp_block_cons<<<sh_idx-ori_idx, THREADS_PER_BLOCK>>>(vertexArray, 
		unsigned int block_num = 13;
		if (sh_idx-ori_idx<block_num) block_num = sh_idx-ori_idx;
		bfs_kernel_dp_block_cons<<<13, THREADS_PER_BLOCK>>>(vertexArray, 
									 	edgeArray, levelArray, buffer+ori_idx, sh_idx-ori_idx, 
										buffer, idx);
	}
	
	// no post work require
}

// recursive BFS traversal with block-level consolidation
__global__ void bfs_kernel_dp_block_malloc_cons(int *vertexArray, int *edgeArray, int *levelArray, 
								unsigned int *queue, unsigned int *buffer, unsigned int *idx) {
	unsigned int bid = blockIdx.x; // 1-Dimensional grid configuration
	unsigned int t_idx;
	__shared__ unsigned int sh_idx;
	__shared__ unsigned int *sh_buffer;
	int node = queue[bid];

	unsigned int num_children = vertexArray[node+1]-vertexArray[node];
	if (threadIdx.x==0) {
		sh_buffer = (unsigned int*)malloc(sizeof(unsigned int)*num_children);
		sh_idx = 0;
	}
	__syncthreads();

	for (unsigned childp = threadIdx.x; childp < num_children; childp+=blockDim.x) {
		int child = edgeArray[vertexArray[node]+childp];
		unsigned node_level = levelArray[node];
		unsigned child_level = levelArray[child];
		if (child_level==UNDEFINED || child_level>(node_level+1)){
			unsigned old_level = atomicMin(&levelArray[child], node_level+1);
			t_idx = atomicInc(&sh_idx, GM_BUFF_SIZE);
			sh_buffer[t_idx] = child;
		}
	}
	__syncthreads();
	if (threadIdx.x==0 && sh_idx>0) {
	//	printf("Launch kernel with %d - %d = %d blocks\n", sh_idx, ori_idx, sh_idx-ori_idx);
		bfs_kernel_dp_block_malloc_cons<<<sh_idx, THREADS_PER_BLOCK>>>(vertexArray, 
									 	edgeArray, levelArray, sh_buffer, 
										buffer, idx);
#ifdef FORCE_SYNC
		hipDeviceSynchronize();
		free(sh_buffer);
#endif
	}
	// no post work require
}

__global__ void dp_grid_cons_init()
{
	tmp_idx = 0;
}

// recursive BFS traversal with grid-level consolidation
// queue and buffer work like Ping-Pong pointer
__global__ void bfs_kernel_dp_grid_cons(int *vertexArray, int *edgeArray, int *levelArray, 
									unsigned int *queue, unsigned int *qidx, 
									unsigned int *buffer, unsigned int *idx,
									unsigned int *count) 
{
	unsigned int bid = blockIdx.x; //+ blockIdx.y*gridDim.x; // 1-Dimensional grid configuration
	unsigned int t_idx;
	__shared__ unsigned int *sh_buffer;
	__shared__ unsigned int sh_idx;
	__shared__ unsigned int ori_idx;
	__shared__ unsigned int offset;
	for ( ; bid<*qidx; bid += gridDim.x ) {
		int node = queue[bid];

		unsigned int num_children = vertexArray[node+1]-vertexArray[node];

		for (unsigned childp = threadIdx.x; childp < num_children; childp+=blockDim.x) {
			int child = edgeArray[vertexArray[node]+childp];
			unsigned node_level = levelArray[node];
			unsigned child_level = levelArray[child];
			if (child_level==UNDEFINED || child_level>(node_level+1)){
				unsigned old_level = atomicMin(&levelArray[child], node_level+1);
				t_idx = atomicInc(idx, GM_BUFF_SIZE);
				//sh_buffer[t_idx] = child;
				buffer[t_idx] = child;
			}
		}
	}
	__syncthreads();

	// 2nd phase, grid level kernel launch
	if (threadIdx.x==0) {
		// count up
		if (atomicInc(count, MAXDIMGRID) >= (gridDim.x-1) ) {
#ifdef GPU_PROFILE
			atomicInc(&nested_calls, INF);
#endif
			//printf("Buffer size %d\n", *idx);
			*count = 0;	// reset counter
			*qidx = 0;	// reset next buffer index
			tmp_idx = 0;
			dim3 dimGrid(1,1,1);
			dimGrid.x = 13 * 16;
			if (*idx<=208) 	dimGrid.x = *idx;

			bfs_kernel_dp_grid_cons<<<dimGrid, THREADS_PER_BLOCK>>>(vertexArray, edgeArray,
								levelArray, buffer, idx, queue, qidx, count);
	
//			bfs_kernel_dp_grid_cons<<<dimGrid, THREADS_PER_BLOCK>>>(vertexArray, edgeArray, 
//								levelArray, buffer+ori_idx, qidx, buffer, idx, count);
#ifdef FORCE_SYNC
			hipDeviceSynchronize();
#endif
		}
	}
	// no post work require
}

// recursive BFS traversal with grid-level consolidation
__global__ void bfs_kernel_dp_grid_malloc_cons(int *vertexArray, int *edgeArray, int *levelArray, 
								unsigned int *queue, unsigned int *qidx, 
								unsigned int *buffer, unsigned int *idx,
								unsigned int *count) 
{
#if (PROFILE_GPU!=0)
	if (threadIdx.x+blockDim.x*blockIdx.x==0) nestd_calls++;
#endif
	unsigned int bid = blockIdx.x; // 1-Dimensional grid configuration
	unsigned int t_idx;
	__shared__ int *sh_buffer;
	__shared__ unsigned int sh_idx;
	//__shared__ unsigned int ori_idx;
	__shared__ unsigned int offset;
	int node = queue[bid];

	unsigned int num_children = vertexArray[node+1]-vertexArray[node];
	if (threadIdx.x==0) {
		sh_buffer = (int*)malloc(sizeof(int)*num_children);
		sh_idx = 0;
		//ori_idx = atomicAdd(idx, num_children);
		//sh_idx = ori_idx;
	}
	__syncthreads();

	for (unsigned childp = threadIdx.x; childp < num_children; childp+=blockDim.x) {
		int child = edgeArray[vertexArray[node]+childp];
		unsigned node_level = levelArray[node];
		unsigned child_level = levelArray[child];
		if (child_level==UNDEFINED || child_level>(node_level+1)){
			unsigned old_level = atomicMin(&levelArray[child], node_level+1);
			t_idx = atomicInc(&sh_idx, GM_BUFF_SIZE);
			sh_buffer[t_idx] = child;
		}
	}
	__syncthreads();
	// reorganize consolidation buffer for load balance ()
	if (threadIdx.x==0) {
		//offset = atomicAdd(qidx, sh_idx-ori_idx);
		offset = atomicAdd(idx, sh_idx);
	}
	__syncthreads();
	// dump block_level buffer to grids
	for (unsigned tid = threadIdx.x; tid<sh_idx; tid+=blockDim.x) {
		int gm_idx = tid + offset;
		buffer[gm_idx] = sh_buffer[tid];
	}
	__syncthreads();

	// 2nd phase, grid level kernel launch
	if (threadIdx.x==0) {
		free(sh_buffer);	// free allocated block buffer
		// count up
		if (atomicInc(count, MAXDIMGRID) >= (gridDim.x-1) && *idx!=0 ) {
#ifdef GPU_PROFILE
			atomicInc(&nested_calls, INF);
#endif
			printf("Buffer size %d\n", *idx);
//			*count = malloc(sizeof(unsigned int));
//			*qidx = malloc(sizeof(unsigned int));
			*count = 0;	// reset counter
			*qidx = 0;	// reset next buffer index
			dim3 dimGrid(1,1,1);
			if (*idx<=MAXDIMGRID) {
				dimGrid.x = *idx;
			}
		/*	else if (*idx<=MAXDIMGRID*THREADS_PER_BLOCK) {
				dimGrid.x = MAXDIMGRID;
				dimGrid.y = *idx/MAXDIMGRID+1;
			}*/
			else {
				printf("Too many elements in queue\n");
			}

			bfs_kernel_dp_grid_malloc_cons<<<dimGrid, THREADS_PER_BLOCK>>>(vertexArray, edgeArray,
								levelArray, buffer, idx, queue, qidx, count);
	
#ifdef FORCE_SYNC
			hipDeviceSynchronize();
			free(sh_buffer);
#endif
		}
	}

//	if (threadIdx.x==0 && sh_idx>ori_idx) {
	//	printf("Launch kernel with %d - %d = %d blocks\n", sh_idx, ori_idx, sh_idx-ori_idx);
//		bfs_kernel_dp_grid_cons<<<sh_idx-ori_idx, THREADS_PER_BLOCK>>>(vertexArray, 
//									 	edgeArray, levelArray, buffer+ori_idx, 
//										buffer, idx);
//	}

	// no post work require
}

// recursive BFS traversal with grid-level consolidation
// queue and buffer work like Ping-Pong pointer
__global__ void bfs_kernel_dp_grid_cons_complex(int *vertexArray, int *edgeArray, int *levelArray, 
									unsigned int *queue, unsigned int *qidx, 
									unsigned int *buffer, unsigned int *idx,
									unsigned int *count) 
{
#if (PROFILE_GPU!=0)
	if (threadIdx.x+blockDim.x*blockIdx.x==0) nestd_calls++;
#endif
	unsigned int bid = blockIdx.x; //+ blockIdx.y*gridDim.x; // 1-Dimensional grid configuration
	unsigned int t_idx;
	__shared__ unsigned int *sh_buffer;
	__shared__ unsigned int sh_idx;
	__shared__ unsigned int ori_idx;
	__shared__ unsigned int offset;
	int node = queue[bid];

	unsigned int num_children = vertexArray[node+1]-vertexArray[node];
	if (threadIdx.x==0) {
		ori_idx = atomicAdd(&tmp_idx, num_children);
		sh_idx = 0;
		sh_buffer = tmp_buffer+ori_idx;
	}
	__syncthreads();

	for (unsigned childp = threadIdx.x; childp < num_children; childp+=blockDim.x) {
		int child = edgeArray[vertexArray[node]+childp];
		unsigned node_level = levelArray[node];
		unsigned child_level = levelArray[child];
		if (child_level==UNDEFINED || child_level>(node_level+1)){
			unsigned old_level = atomicMin(&levelArray[child], node_level+1);
			t_idx = atomicInc(&sh_idx, GM_BUFF_SIZE);
			sh_buffer[t_idx] = child;
		}
	}
	__syncthreads();
	// reorganize consolidation buffer for load balance ()
	if (threadIdx.x==0) {
		//offset = atomicAdd(qidx, sh_idx-ori_idx);
		offset = atomicAdd(idx, sh_idx);
	}
	__syncthreads();
	// dump block-level buffer to grid-level buffer
	for (unsigned tid = threadIdx.x; tid<sh_idx; tid+=blockDim.x) {
		int gm_idx = tid + offset;
		buffer[gm_idx] = sh_buffer[tid];
	}
	__syncthreads();

	// 2nd phase, grid level kernel launch
	if (threadIdx.x==0) {
		// count up
		if (atomicInc(count, MAXDIMGRID) >= (gridDim.x-1) ) {
#ifdef GPU_PROFILE
			atomicInc(&nested_calls, INF);
#endif
			//printf("Buffer size %d\n", *idx);
			*count = 0;	// reset counter
			*qidx = 0;	// reset next buffer index
			tmp_idx = 0;
			dim3 dimGrid(1,1,1);
			if (*idx<=MAXDIMGRID) {
				dimGrid.x = *idx;
			}
			/*else if (*idx<=MAXDIMGRID*THREADS_PER_BLOCK) {
				dimGrid.x = MAXDIMGRID;
				dimGrid.y = *idx/MAXDIMGRID+1;
			}*/
			else {
				printf("%d \n", *idx);
				printf("Too many elements in queue\n");
			}

			bfs_kernel_dp_grid_cons<<<dimGrid, THREADS_PER_BLOCK>>>(vertexArray, edgeArray,
								levelArray, buffer, idx, queue, qidx, count);
	
//			bfs_kernel_dp_grid_cons<<<dimGrid, THREADS_PER_BLOCK>>>(vertexArray, edgeArray, 
//								levelArray, buffer+ori_idx, qidx, buffer, idx, count);
#ifdef FORCE_SYNC
			hipDeviceSynchronize();
#endif
		}
	}
	// no post work require
}


#endif
