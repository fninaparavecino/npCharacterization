#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "bfs_rec.h"

#define QMAXLENGTH 10240000*10
#define GM_BUFF_SIZE 10240000*10

#ifndef THREADS_PER_BLOCK_FLAT	//block size for flat parallelism
#define THREADS_PER_BLOCK_FLAT 128
#endif

#ifndef NUM_BLOCKS_FLAT
#define NUM_BLOCKS_FLAT 256
#endif

#ifndef THREADS_PER_BLOCK // nested kernel block size
#define THREADS_PER_BLOCK 64
#endif

#ifndef CONSOLIDATE_LEVEL
#define CONSOLIDATE_LEVEL 0
#endif

#define STREAMS 4

#include "bfs_rec_kernel.cu"

int *d_vertexArray;
int *d_edgeArray;
int *d_levelArray;
int *d_work_queue;
char *d_frontier;
char *d_update;

unsigned int *d_queue_length;
unsigned int *d_nonstop;

dim3 dimGrid(1,1,1);	// thread+bitmap
dim3 dimBlock(1,1,1);

//char *update = new char [noNodeTotal] ();
//int *queue = new int [queue_max_length];
unsigned int queue_max_length = QMAXLENGTH;
unsigned int queue_length = 0;
unsigned int nonstop = 0;

inline void cudaCheckError(const char* file, int line, hipError_t ce)
{
	if (ce != hipSuccess){
		printf("Error: file %s, line %d %s\n", file, line, hipGetErrorString(ce));
		exit(1);
	}
}

void prepare_gpu()
{
	start_time = gettime_ms();
	hipFree(NULL);
	end_time = gettime_ms();
	init_time += end_time - start_time;

	if (DEBUG) {
		fprintf(stderr, "Choose CUDA device: %d\n", config.device_num);
		fprintf(stderr, "hipSetDevice:\t\t%lf\n",end_time-start_time);
	}

	start_time = gettime_ms();
	size_t limit = 0;
	if (DEBUG) {
		cudaCheckError( __FILE__, __LINE__, hipDeviceGetLimit(&limit, hipLimitMallocHeapSize));
		printf("cudaLimistMallocHeapSize: %u\n", (unsigned)limit);
	}
	limit = 102400000;
	cudaCheckError( __FILE__, __LINE__, hipDeviceSetLimit(hipLimitMallocHeapSize, limit));
	if (DEBUG) {
		cudaCheckError( __FILE__, __LINE__, hipDeviceGetLimit(&limit, hipLimitMallocHeapSize));
		printf("cudaLimistMallocHeapSize: %u\n", (unsigned)limit);
	}
	end_time = gettime_ms();
	//fprintf(stderr, "Set Heap Size:\t\t%.2lf ms.\n", end_time-start_time);

	/* Allocate GPU memory */
	start_time = gettime_ms();
	cudaCheckError( __FILE__, __LINE__, hipMalloc( (void**)&d_vertexArray, sizeof(int)*(noNodeTotal+1) ) );
	cudaCheckError( __FILE__, __LINE__, hipMalloc( (void**)&d_edgeArray, sizeof(int)*noEdgeTotal ) );
	cudaCheckError( __FILE__, __LINE__, hipMalloc( (void**)&d_levelArray, sizeof(int)*noNodeTotal ) );
	printf("DEBUG levelArray : %d \n", noNodeTotal);
	//cudaCheckError( __LINE__, hipMalloc( (void**)&d_nonstop, sizeof(unsigned int) ) );
	end_time = gettime_ms();
	d_malloc_time += end_time - start_time;

	start_time = gettime_ms();
	cudaCheckError( __FILE__, __LINE__, hipMemcpy( d_vertexArray, graph.vertexArray, sizeof(int)*(noNodeTotal+1), hipMemcpyHostToDevice) );
	cudaCheckError( __FILE__, __LINE__, hipMemcpy( d_edgeArray, graph.edgeArray, sizeof(int)*noEdgeTotal, hipMemcpyHostToDevice) );
	//copy the level array from CPU to GPU
	cudaCheckError( __FILE__, __LINE__, hipMemcpy( d_levelArray, graph.levelArray, sizeof(int)*noNodeTotal, hipMemcpyHostToDevice) );
	end_time = gettime_ms();
	h2d_memcpy_time += end_time - start_time;
}

void clean_gpu()
{
	hipFree(d_vertexArray);
	hipFree(d_edgeArray);
	hipFree(d_levelArray);
}

// ----------------------------------------------------------
// version #1 - flat parallelism - level-based BFS traversal
// ----------------------------------------------------------

void bfs_flat_gpu()
{
	/* prepare GPU */

	bool queue_empty = false;
	bool *d_queue_empty;

	cudaCheckError(  __FILE__, __LINE__, hipMalloc( &d_queue_empty, sizeof(bool)) );
  printf("Grid configuration gridxblocks, %d x %d\n", NUM_BLOCKS_FLAT, THREADS_PER_BLOCK_FLAT);
	unsigned level = 0;

	//level-based traversal
	while (!queue_empty){
		cudaCheckError(  __FILE__, __LINE__, hipMemset( d_queue_empty, true, sizeof(bool)) );

		bfs_kernel_flat<<<1, 32>>>(level,noNodeTotal, d_vertexArray, d_edgeArray, d_levelArray, d_queue_empty);
		cudaCheckError(  __FILE__, __LINE__, hipGetLastError());
		cudaCheckError(  __FILE__, __LINE__, hipMemcpy( &queue_empty, d_queue_empty, sizeof(bool), hipMemcpyDeviceToHost) );
		level++;
	}

	if (DEBUG)
		printf("===> GPU #1 - flat parallelism.\n");

}

// ----------------------------------------------------------
// version #2 - dynamic parallelism - naive
// ----------------------------------------------------------
void bfs_rec_dp_naive_gpu()
{
	/* prepare GPU */

	int children = graph.vertexArray[source+1]-graph.vertexArray[source];
	unsigned block_size = min (children, THREADS_PER_BLOCK);
	bfs_kernel_dp<<<1,block_size>>>(source, d_vertexArray, d_edgeArray, d_levelArray);
	cudaCheckError(  __FILE__, __LINE__, hipGetLastError());
	cudaCheckError(  __FILE__, __LINE__, hipDeviceSynchronize());

	if (DEBUG)
		printf("===> GPU #2 - nested parallelism naive.\n");
}

// ----------------------------------------------------------
// version #3 - dynamic parallelism - hierarchical
// ----------------------------------------------------------
void bfs_rec_dp_hier_gpu()
{
	//recursive BFS traversal - hierarchical
	int children = graph.vertexArray[source+1]-graph.vertexArray[source];
	bfs_kernel_dp_hier<<<children, THREADS_PER_BLOCK>>>(source, d_vertexArray, d_edgeArray, d_levelArray);
	cudaCheckError(  __FILE__, __LINE__, hipGetLastError());
	cudaCheckError(  __FILE__, __LINE__, hipDeviceSynchronize());
	if (DEBUG)
		printf("===> GPU #3 - nested parallelism hierarchical.\n", gettime_ms()-start_time);
}

// ----------------------------------------------------------
// version #4 - dynamic parallelism - consolidation
// ----------------------------------------------------------
void bfs_rec_dp_cons_gpu()
{
	//recursive BFS traversal - dynamic parallelism consolidation
	unsigned int *d_buffer;
	unsigned int *d_idx;
	cudaCheckError(  __FILE__, __LINE__, hipMalloc( &d_buffer, sizeof(unsigned int)*GM_BUFF_SIZE) );
	cudaCheckError(  __FILE__, __LINE__, hipMalloc( &d_idx, sizeof(unsigned int)) );
    bfs_kernel_dp_cons_prepare<<<1,1>>>(d_levelArray, d_buffer, d_idx, source);

	int children = 1;
	switch (config.solution) {
	case 3:
		if (DEBUG)
			fprintf(stdout, "warp level consolidation\n");
		bfs_kernel_dp_warp_cons<<<children, THREADS_PER_BLOCK>>>(d_vertexArray, d_edgeArray, d_levelArray,
												d_buffer, children, d_buffer, d_idx);
		//bfs_kernel_dp_warp_cons_back<<<children, THREADS_PER_BLOCK>>>(d_vertexArray, d_edgeArray, d_levelArray,
		//										d_buffer, d_buffer, d_idx);
		//bfs_kernel_dp_warp_malloc_cons<<<children, THREADS_PER_BLOCK>>>(d_vertexArray, d_edgeArray, d_levelArray,
		//										d_buffer, d_buffer, d_idx);
		break;
	case 4:
		if (DEBUG)
			fprintf(stdout, "block level consolidation\n");
		bfs_kernel_dp_block_cons<<<children, THREADS_PER_BLOCK>>>(d_vertexArray, d_edgeArray, d_levelArray,
												d_buffer, children, d_buffer, d_idx);
		//bfs_kernel_dp_block_malloc_cons<<<children, THREADS_PER_BLOCK>>>(d_vertexArray, d_edgeArray, d_levelArray,
		//										d_buffer, d_buffer, d_idx);
		break;
	case 5:
		// queue and buffer are different
		// buffer stores the active working set
		unsigned int *d_queue;
		unsigned int *d_qidx;
		unsigned int *d_count;
		cudaCheckError(  __FILE__, __LINE__, hipMalloc( &d_queue, sizeof(unsigned int)*GM_BUFF_SIZE) );
		cudaCheckError(  __FILE__, __LINE__, hipMalloc( &d_qidx, sizeof(unsigned int)) );
		cudaCheckError(  __FILE__, __LINE__, hipMalloc( &d_count, sizeof(unsigned int)) );
		cudaCheckError(  __FILE__, __LINE__, hipMemset( d_qidx, 0, sizeof(unsigned int)) );
		cudaCheckError(  __FILE__, __LINE__, hipMemset( d_count, 0, sizeof(unsigned int)) );
		if (DEBUG)
    		fprintf(stdout, "grid level consolidation\n");
		// by default, it utilize malloc
		dp_grid_cons_init<<<1,1>>>();
		bfs_kernel_dp_grid_cons<<<children, THREADS_PER_BLOCK>>>(d_vertexArray, d_edgeArray, d_levelArray,
												d_buffer, d_idx, d_queue, d_qidx, d_count);
		/*	bfs_kernel_dp_grid_malloc_cons<<<children, THREADS_PER_BLOCK>>>(d_vertexArray, d_edgeArray, d_levelArray,
												d_buffer, d_idx, d_queue, d_qidx, d_count);
		*/
		break;
	default:
		printf("Unsopported solutions\n");
		exit(0);
	}
	hipDeviceSynchronize();
	cudaCheckError(  __FILE__, __LINE__, hipGetLastError());
	cudaCheckError(  __FILE__, __LINE__, hipDeviceSynchronize());

	if (DEBUG)
		printf("===> GPU #4 - nested parallelism consolidation.\n", end_time-start_time);
	//gpu_print<<<1,1>>>(d_idx);
	cudaCheckError( __FILE__, __LINE__, hipFree(d_buffer) );
	cudaCheckError( __FILE__, __LINE__, hipFree(d_idx) );
#if (CONSOLIDATE_LEVEL==2)
	cudaCheckError( __FILE__, __LINE__, hipFree(d_queue) );
	cudaCheckError( __FILE__, __LINE__, hipFree(d_qidx) );
	cudaCheckError( __FILE__, __LINE__, hipFree(d_count) );
#endif
}

void BFS_REC_GPU()
{
	cudaCheckError( __FILE__, __LINE__, hipSetDevice(config.device_num) );
	cudaCheckError( __FILE__, __LINE__, hipDeviceReset());
	prepare_gpu();

#ifdef GPU_PROFILE
	reset_gpu_statistics<<<1,1>>>();
	hipDeviceSynchronize();
#endif

	start_time = gettime_ms();
	switch (config.solution) {
		case 0:  bfs_flat_gpu();	//
			break;
		case 1:  bfs_rec_dp_naive_gpu();	//
			break;
		case 2:  bfs_rec_dp_hier_gpu();	//
			break;
		case 3:
		case 4:
		case 5:  bfs_rec_dp_cons_gpu();	//
			break;
		default:
			break;
	}
	cudaCheckError( __FILE__, __LINE__, hipDeviceSynchronize() );
	end_time = gettime_ms();
	ker_exe_time += end_time - start_time;
#ifdef GPU_PROFILE
	gpu_statistics<<<1,1>>>(config.solution);
	hipDeviceSynchronize();
#endif
	//copy the level array from GPU to CPU;
	start_time = gettime_ms();
	cudaCheckError(  __FILE__, __LINE__, hipMemcpy( graph.levelArray, d_levelArray, sizeof(unsigned)*noNodeTotal, hipMemcpyDeviceToHost) );
	end_time = gettime_ms();
	d2h_memcpy_time += end_time - start_time;

	clean_gpu();
}
