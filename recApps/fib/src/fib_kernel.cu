// fib parallel

#include <hip/hip_runtime.h>
#include <stdio.h>

#define cudaErrCheck(ans) { cudaAssert((ans), __FILE__, __LINE__); }

inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void fib_kernel_plain(int n, unsigned long int* vFib){
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if (tid > n/32)
        return;

    if (n == 0 || n == 1){
        return;
    }
    //printf("===KERNEL=== ThreadIdx: %d\n", tid);
    for(int i=tid*32 + 2; i <= n && i < tid*32 + 32; i++){
        vFib[i] = vFib[i-1] + vFib[i-2];
        //printf("===KERNEL=== fib of %d: %ld\n", i, vFib[i]);
    }
}

void fibGPU(int n, unsigned long int* arrayN)
{
  // time variables
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);

  // device arrayN
  unsigned long int *devArrayN = 0;

  // define device
  cudaErrCheck(hipSetDevice(0));

  // cuda malloc for devArrayN
  cudaErrCheck(hipMalloc((void**)&devArrayN, sizeof(unsigned long int)*(n+1)));

  // cuda memcopy
  cudaErrCheck(hipMemcpy(devArrayN, arrayN, sizeof(unsigned long int)*(n+1), hipMemcpyHostToDevice));

  // call the kernel
  dim3 threadsPerBlock(32, 1, 1);
  dim3 blocksPerGrid((n+31)/32, 1, 1);

  printf("Launching fib_kernel (%d x %d)...\n", blocksPerGrid.x, threadsPerBlock.x);
  hipEventRecord(start, 0);
  fib_kernel_plain<<<blocksPerGrid, threadsPerBlock>>>(n, devArrayN);
  cudaErrCheck(hipDeviceSynchronize());
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  //Display time
	hipEventElapsedTime(&time, start, stop);
	printf("\tParallel Kernel time: %.2f ms\n", time);

  // retrieve results
  cudaErrCheck(hipMemcpy(arrayN, devArrayN, sizeof(unsigned long int)*(n+1), hipMemcpyDeviceToHost));

  //Free resource
  hipFree(devArrayN);
}
