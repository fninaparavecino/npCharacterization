#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "prim.h"

#ifndef THREADS_PER_BLOCK // nested kernel block size
#define THREADS_PER_BLOCK 64
#endif

#ifndef WARP_SIZE // nested kernel block size
#define WARP_SIZE 32
#endif

int *d_vertexArray;
int *d_edgeArray;
int *d_levelArray;
bool *d_visitedArray;
int *d_keyArray;
int *d_weightArray;
int *d_nodesVisited;

#define cudaErrCheck(ans) { cudaAssert((ans), __FILE__, __LINE__); }

inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
/***********************************************************
        Prim Recursive version 0
************************************************************/
__global__ void primRec(int node, int numNodes, int* vertexArray, int* edgeArray,
                        int* weightArray, bool* visitedArray, int* keyArray, int* mstParent){
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if (idx > numNodes)
    return;

  // mark node as visited
  visitedArray[node] = true;

  int child = edgeArray[vertexArray[node] + idx];
  int childId = vertexArray[node] + idx;
  int childWeight = weightArray[vertexArray[node] + idx];

  // set min to first child
  int minChild = edgeArray[vertexArray[node]];
  int minChildId = vertexArray[node];
  int minChildWeight = weightArray[vertexArray[node]];

  // find minChild among children
  for (int i=1;i < blockDim.x && i < WARP_SIZE; i++){
    int childShfl = __shfl(child, i);
    int childIdShfl = __shfl(childId, i);
    int weightShfl = __shfl(childWeight, i);

    if (weightShfl < minChildWeight){
      minChildWeight = weightShfl;
      minChild = childShfl;
      minChildId = childIdShfl;
    }
  }

  // if Child explored is different than childSelected
  if (child != minChild)
    return;

  if (visitedArray[minChild] == false && weightArray[minChildId] < keyArray[minChild]){
    //printf("===GPU Kernel=== child selected: %d\n", minChild);
    mstParent[minChild] = node;
    keyArray[minChild] = weightArray[minChildId];
    int grandChildren = vertexArray[minChild+1] - vertexArray[minChild];
    primRec<<<1, grandChildren>>>(minChild, numNodes, vertexArray, edgeArray, weightArray, visitedArray, keyArray, mstParent);
  }
}
/***********************************************************
        Prim Recursive version 1
************************************************************/
__global__ void primPhase2(int node, int numChildren, int* vertexArray, int* edgeArray,
                        int* weightArray, bool* visitedArray, int* keyArray, int* mstParent){
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if (idx > numChildren)
    return;

  int child = edgeArray[vertexArray[node] + idx];
  int childId = vertexArray[node] + idx;

  if (visitedArray[child] == false && weightArray[childId] < keyArray[child]){
    //printf("===GPU Kernel=== child selected: %d\n", minChild);
    mstParent[child] = node;
    keyArray[child] = weightArray[childId];
    int grandChildren = vertexArray[child+1] - vertexArray[child];
    primRec<<<1, grandChildren>>>(child, grandChildren, vertexArray, edgeArray, weightArray, visitedArray, keyArray, mstParent);
  }
}

__global__ void primPhase1(int numNodes, int* nodesVisited, int* vertexArray, int* edgeArray,
                        int* weightArray, bool* visitedArray, int* keyArray, int* mstParent){
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if (idx > numNodes)
    return;

  int node = idx;
  int nodeKey = keyArray[node];

  int minNode = 0;
  int minNodeKey = keyArray[0];

  //while(nodesVisited[0] < numNodes){
    // find minChild among children
    for (int i=1; i < numNodes && i < WARP_SIZE; i++){
      int nodeShfl = __shfl(node, i);
      int keyShfl = __shfl(nodeKey, i);

      if (visitedArray[keyShfl] == false && keyShfl < minNodeKey){
        minNodeKey = keyShfl;
        minNode = nodeShfl;
      }
    }

    if(minNode != node)
      return;

    // mark node as visited
    visitedArray[minNode] = true;
    atomicAdd(&nodesVisited[0], 1);
    __syncthreads();
    int children = vertexArray[minNode+1] - vertexArray[minNode];
    primPhase2<<<1, children>>>(minNode, children, vertexArray, edgeArray,
                             weightArray, visitedArray, keyArray, mstParent);

  //}
}
// ----------------------------------------------------------
// Implementation 0: Recursive MST using Prim's algorithm
// ----------------------------------------------------------
void primRecWrapper()
{
	hipEvent_t start, stop;
	float time;
	/* prepare GPU */

	int children = graph.vertexArray[source+1]-graph.vertexArray[source];
	unsigned block_size = min (children, THREADS_PER_BLOCK);
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	primRec<<<1,block_size>>>(source, noNodeTotal, d_vertexArray, d_edgeArray, d_weightArray, d_visitedArray, d_keyArray, d_levelArray);
	cudaErrCheck( hipGetLastError());
	cudaErrCheck( hipDeviceSynchronize());
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	//Display time
	hipEventElapsedTime(&time, start, stop);
	printf("\tParallel Job time: %.2f ms\n", time);

	if (DEBUG)
		printf("===> GPU Prim rec.\n");
}

// ----------------------------------------------------------
// Implementation 1: Prim's algorithm using 2 phases
// ----------------------------------------------------------
void primWrapper2Phases()
{
  hipEvent_t start, stop;
  float time;

  // prepare GPU
  unsigned block_size = min (noNodeTotal, THREADS_PER_BLOCK);

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);
  primPhase1<<<1, block_size>>>(noNodeTotal, d_nodesVisited, d_vertexArray, d_edgeArray, d_weightArray, d_visitedArray, d_keyArray, d_levelArray);
  cudaErrCheck( hipGetLastError());
  cudaErrCheck( hipDeviceSynchronize());
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  //Display time
  hipEventElapsedTime(&time, start, stop);
  printf("\tParallel Job time: %.2f ms\n", time);

  if (DEBUG)
  	printf("===> GPU Prim rec.\n");
}
// ----------------------------------------------------------
// Implementation 2:
// ----------------------------------------------------------
void primWrapperControl()
{
  hipEvent_t start, stop;
  float time;

  // prepare GPU
  int children = graph.vertexArray[source+1] - graph.vertexArray[source];

  unsigned block_size = min (children, THREADS_PER_BLOCK);

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  int minNode = source;
  int minKey = graph.keyArray[source];

  while(graph.nodesVisited[0] < noNodeTotal){
    for(int i=0; i< noNodeTotal; i++){
      if (graph.visited[i] == false && graph.keyArray[i] < minKey){
        minNode = i;
        minKey = graph.keyArray[i];
      }
    }

    primRec<<<1, block_size>>>(minNode, noNodeTotal, d_vertexArray, d_edgeArray, d_weightArray, d_visitedArray, d_keyArray, d_levelArray);
    cudaErrCheck( hipDeviceSynchronize());
    cudaErrCheck( hipMemcpy( graph.visited, d_visitedArray, sizeof(char)*noNodeTotal, hipMemcpyDeviceToHost) );
    cudaErrCheck( hipMemcpy( graph.keyArray, d_keyArray, sizeof(int)*noNodeTotal, hipMemcpyDeviceToHost) );
    cudaErrCheck( hipMemcpy( graph.nodesVisited, d_nodesVisited, sizeof(int)*1, hipMemcpyDeviceToHost) );
  }

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  //Display time
  hipEventElapsedTime(&time, start, stop);
  printf("\tParallel Job time: %.2f ms\n", time);

  if (DEBUG)
  	printf("===> GPU Prim rec.\n");
}
void prepare_gpu()
{
	start_time = gettime_ms();
	hipFree(NULL);
	end_time = gettime_ms();
	init_time += end_time - start_time;

	if (DEBUG) {
		fprintf(stderr, "Choose CUDA device: %d\n", config.device_num);
		fprintf(stderr, "hipSetDevice:\t\t%lf\n",end_time-start_time);
	}

	start_time = gettime_ms();
	size_t limit = 0;
	if (DEBUG) {
		cudaErrCheck(hipDeviceGetLimit(&limit, hipLimitMallocHeapSize));
		printf("cudaLimistMallocHeapSize: %u\n", (unsigned)limit);
	}
	limit = 102400000;
	cudaErrCheck( hipDeviceSetLimit(hipLimitMallocHeapSize, limit));
	if (DEBUG) {
		cudaErrCheck(hipDeviceGetLimit(&limit, hipLimitMallocHeapSize));
		printf("cudaLimistMallocHeapSize: %u\n", (unsigned)limit);
	}
	end_time = gettime_ms();

	/* Allocate GPU memory */
	start_time = gettime_ms();
	cudaErrCheck(hipMalloc( (void**)&d_vertexArray, sizeof(int)*(noNodeTotal+1) ) );
	cudaErrCheck(hipMalloc( (void**)&d_edgeArray, sizeof(int)*noEdgeTotal ) );
	cudaErrCheck(hipMalloc( (void**)&d_levelArray, sizeof(int)*noNodeTotal ) );
  cudaErrCheck(hipMalloc( (void**)&d_visitedArray, sizeof(bool)*noNodeTotal ) );
  cudaErrCheck(hipMalloc( (void**)&d_weightArray, sizeof(int)*noEdgeTotal ) );
  cudaErrCheck(hipMalloc( (void**)&d_keyArray, sizeof(int)*noNodeTotal ) );
  cudaErrCheck(hipMalloc( (void**)&d_nodesVisited, sizeof(int)*1 ) );

	end_time = gettime_ms();
	d_malloc_time += end_time - start_time;

	start_time = gettime_ms();
	cudaErrCheck( hipMemcpy( d_vertexArray, graph.vertexArray, sizeof(int)*(noNodeTotal+1), hipMemcpyHostToDevice) );
	cudaErrCheck( hipMemcpy( d_edgeArray, graph.edgeArray, sizeof(int)*noEdgeTotal, hipMemcpyHostToDevice) );
	cudaErrCheck( hipMemcpy( d_levelArray, graph.levelArray, sizeof(int)*noNodeTotal, hipMemcpyHostToDevice) );
  cudaErrCheck( hipMemcpy( d_visitedArray, graph.visited, sizeof(bool)*noNodeTotal, hipMemcpyHostToDevice) );
  cudaErrCheck( hipMemcpy( d_weightArray, graph.weightArray, sizeof(int)*noEdgeTotal, hipMemcpyHostToDevice) );
  cudaErrCheck( hipMemcpy( d_keyArray, graph.keyArray, sizeof(int)*noNodeTotal, hipMemcpyHostToDevice) );
  cudaErrCheck( hipMemcpy( d_nodesVisited, graph.nodesVisited, sizeof(int)*1, hipMemcpyHostToDevice) );
	end_time = gettime_ms();
	h2d_memcpy_time += end_time - start_time;
}
void clean_gpu()
{
	hipFree(d_vertexArray);
	hipFree(d_edgeArray);
	hipFree(d_levelArray);
  hipFree(d_visitedArray);
	hipFree(d_weightArray);
  hipFree(d_keyArray);
  hipFree(d_nodesVisited);
}

void primGPU()
{
	cudaErrCheck( hipSetDevice(config.device_num) );
	cudaErrCheck( hipDeviceReset());
	prepare_gpu();

#ifdef GPU_PROFILE
	reset_gpu_statistics<<<1,1>>>();
	hipDeviceSynchronize();
#endif

	start_time = gettime_ms();
	switch (config.solution) {
		case 0: primRecWrapper();	//GPU rec implementation
			break;
    case 1: primWrapper2Phases(); // Prim GPU with 2 phases
      break;
		default:
      printf("===ERROR=== Solution selected not available\n");
			break;
	}
	cudaErrCheck(hipDeviceSynchronize() );
	end_time = gettime_ms();
	ker_exe_time += end_time - start_time;
#ifdef GPU_PROFILE
	gpu_statistics<<<1,1>>>(config.solution);
	hipDeviceSynchronize();
#endif
	//copy the level array from GPU to CPU;
	start_time = gettime_ms();
	cudaErrCheck( hipMemcpy( graph.levelArray, d_levelArray, sizeof(unsigned)*noNodeTotal, hipMemcpyDeviceToHost) );
	end_time = gettime_ms();
	d2h_memcpy_time += end_time - start_time;

	clean_gpu();
}
